
#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>

// CUDA Kernel function to add the elements of two arrays on the GPU
__global__
void add(int n, float* x, float* y) {
	int index = blockIdx.x * blockDim.x + threadIdx.x; // index of current thread within its block
	int stride = gridDim.x * blockDim.x; // number of threads in block
	for (int i = index; i < n; i += stride) {
		y[i] += x[i];
	}
}

int main(void) {
	int N = 1<<20; // 1M elements

	float *x, *y;

	// allocation unifed memory -- accessible from CPU and GPU
	hipMallocManaged(&x, N*sizeof(float));
	hipMallocManaged(&y, N*sizeof(float));

	for (int i = 0; i < N; i++) {
		y[i] = 1.f;
		x[i] = 2.f;
	}
	// launching add() kernel, which invokes on the GPU
	// <<<n, m>>> -- m is number of threads in a thread block, n is number of
	// thread blocks;
	int blockSize = 256;
	int numBlocks = (N + blockSize - 1) / blockSize;
	add<<<numBlocks, 256>>>(N, x, y);


	// making CPU to wait until the kernel is done
	hipDeviceSynchronize();
	float maxError = 0.f;
	for (int i = 0; i < N; i++)
		maxError = std::fmax(maxError, std::fabs(y[i] - 3.f));

	std::cout << "Max error: " << maxError << std::endl;

	// free memory
	hipFree(x);
	hipFree(y);

	return 0;
}
