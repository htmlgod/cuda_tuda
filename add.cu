
#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>

// CUDA Kernel function to add the elements of two arrays on the GPU
__global__
void add(int n, float* x, float* y) {
	for (int i = 0; i < n; i++) {
		y[i] += x[i];
	}
}

int main(void) {
	int N = 1<<20; // 1M elements

	float *x, *y;

	// allocation unifed memory -- accessible from CPU and GPU
	hipMallocManaged(&x, N*sizeof(float));
	hipMallocManaged(&y, N*sizeof(float));

	for (int i = 0; i < N; i++) {
		y[i] = 1.f;
		x[i] = 2.f;
	}
	// launching add() kernel, which invokes on the GPU
	add<<<1, 1>>>(N, x, y);


	// making CPU to wait until the kernel is done
	hipDeviceSynchronize();
	float maxError = 0.f;
	for (int i = 0; i < N; i++)
		maxError = std::fmax(maxError, std::fabs(y[i] - 3.f));

	std::cout << "Max error: " << maxError << std::endl;

	// free memory
	hipFree(x);
	hipFree(y);

	return 0;
}
